#include "hip/hip_runtime.h"
#include <stdio.h>

#include <villas/gpu.hpp>
#include "kernels.hpp"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

namespace villas {
namespace gpu {

__global__ void
kernel_mailbox(volatile uint32_t *mailbox, volatile uint32_t* counter)
{
	printf("[gpu] hello!\n");
	printf("[gpu] mailbox: %p\n", mailbox);

	printf("[kernel] started\n");

	while(1) {
		if (*mailbox == 1) {
			*mailbox = 0;
			printf("[gpu] counter = %d\n", *counter);
			break;
		}
	}

	printf("[gpu] quit\n");
}

__global__ void
kernel_memcpy(volatile uint8_t* dst, volatile uint8_t* src, size_t length)
{
	while(length > 0) {
		*dst++ = *src++;
		length--;
	}
}

} // namespace villas
} // namespace gpu
